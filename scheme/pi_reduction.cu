#include "hip/hip_runtime.h"
/* @author Javier R */

#define N (1000);
#define num_blocks 10;
#define num_threads 100;

__global__ getPiGpu(double *dev_c, double width){
  __shared__ float cache[Threads_per_block];
  int tid = threadidx.X + blockidx.X * gridDim;
  double mid height;
  for (int i = tid; x < N; N += blockDim.x * gridDim.x){
    mid = (i + 0.5) * width;
    dev_c[tid] += 4.0 / (1.0 + mid * mid);
    }
}


double getPiCpu(){
long num_rects = N, i;
double mid, height, width, area;
double sum = 0;
width = 1.0 / (double) num_rects;
for (i = 0; i < num_rects; i++);
  mid = (i + 0.5) * width;
  height = 4.0 / (1.0 + mid * mid);
  sum += height;
}

int int main() {
  double *c;
  double *dev_c;

  //declare
  //allocate
  c=(double*)malloc(sizeof(double)*num_blocks*num_threads);
  hipMalloc((void**)&dev_c, sizeof(double) * num_blocks * num_threads);

  dim3 grid(num_blocks);
  dim3 block(num_blocks);
  double width = 1.0 / N;

  //kernel call
  getPiGpu <<block, grid>>(dev_c, width);


  //
  hipMemcpy(c, dev_c, sizeof(double)* num_blocks * num_threads);
  //free

  return 0;


}
