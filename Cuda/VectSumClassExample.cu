#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N (4096*4096)
#define THREADS_PER_BLOCK 512

__global__ void sumOnGpu(int *a, int *b, int *c, int n){
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	if(index < n){
		c[index] = a[index] + b[index];
	}
}

void sumOnCpu(int *a, int *b, int *c, int n){
	for(int i = 0; i < n; i++){
		c[i] = a[i] + b[i];
	}
}

void randomNumbers(int *a, int n){
	for(int i = 0; i < n; i++){
		a[i] = rand()%100000;
	}
}

int compare(int *a, int *b, int n){
	int pass = 1;
	for(int i = 0; i < n; i++){
		if(a[i] != b[i]){
			printf("Different values at a[%i] = %i and b[%i] = %i \n", i, a[i], i, b[i]);
			pass = 0;
		}
	}
	if (pass)
		printf("args are the same \n");
	else
		printf("args are different \n");
	return pass;
}


int main(){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	a = (int*)malloc(size);
	b = (int*)malloc(size);
	c = (int*)malloc(size);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	randomNumbers(a, N);
	randomNumbers(b, N);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	clock_t timeOnGpu = clock();

	sumOnGpu<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);

	printf("time on GPU %f \n", ((double)clock() - timeOnGpu)/CLOCKS_PER_SEC);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	int* c_h;
	c_h = (int*)malloc(size);

	clock_t i = clock();

	sumOnCpu(a, b, c_h, N);
	printf("time on CPU %f \n", ((double)clock() - i)/CLOCKS_PER_SEC);

	compare(c, c_h, N);
	free(a);
	free(b);
	free(c);
	free(c_h);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;

}
