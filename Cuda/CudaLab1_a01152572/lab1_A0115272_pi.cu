/*
    Author Javier Rodríguez
           A01152572
*/

#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#define N (1000000)
#define THREADS_PER_BLOCK 1000

//pi on cpu
double getPiCpu(){
  long num_rects = N, i;
  double mid, height, width, area;
  double sum = 0.0;

  width = 1.0 / (double) num_rects;

  for (i = 0; i < num_rects; i++) {
    mid = (i + 0.5) * width;
    height = 4.0 / (1.0 + mid * mid);
    sum += height;
  }
  area = width * sum;
  return area;

}

//Pi gpu
__global__ void getPiGpu(double *a, long n) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  //Pi variables
  double mid, width;

  width = 1.0 / (long) n;

        if(tid < n){
          mid = (tid + 0.5) * width;
          a[tid] = 4.0 / (1.0 + mid * mid);
          }
      }

      // double piSum(double *a){
      //   double sum, width, pi;
      //   long num_rects = N;
      //
      //   width = 1.0 / (double) num_rects;
      //
      //   for (long i = 0; i < N; i++) {
      //       sum += a[i];
      //   }
      //     pi = width * sum;
      //     return pi;
      // }

      int main() {
        double piCpu, piGpu;
        double sum, width;
        double a[N];
        double *d_a;
        double size = N * sizeof(double);

        d_a=(double*)malloc(size);
        hipMalloc((void**)&d_a, size);
        //time on gpu
        clock_t timeOnGpu = clock();
        //kernel call
        getPiGpu<<<N/THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, N);
        //devicetohost recuperar array de heights
        hipMemcpy(a, d_a, size, hipMemcpyDeviceToHost);
        hipFree(d_a);
        // piGpu = piSum(a[N]);
        width = 1.0 / (double) N;

        for (long i = 0; i < N; i++) {
           sum += a[i];
        }

        piGpu = width * sum;

        printf("%f\n", piGpu);
                printf("time on GPU %f \n", ((double)clock() - timeOnGpu)/CLOCKS_PER_SEC);

        //Get pi cpu and print
        clock_t timeOnCpu = clock();
        piCpu = getPiCpu();
        printf("%lf\n", piCpu);
        printf("time on CPU %f \n", ((double)clock() - timeOnCpu)/CLOCKS_PER_SEC);
                return 0;
        }
