/*
•       Implement a matrix multiplication using a GPU to solve the operation instead of a CPU. Create the matrices in the CPU pass them to the GPU calculate the answer and show the answer in the console in an ordered way.
*/
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>  // rand(), srand()
#include <time.h>

__global__ void gpuMatrix(int *m1, int *m2, int *m3, int dim) {
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  int temp = 0;

  int row = tid/dim;
  int col = tid % dim;

  if(col < dim && row < dim){
    for (int i = 0; i < dim; i++) {
      temp += m1[row*dim+i]*m2[i*dim+col];
    }
   m3[tid] = temp;
  }
}
// void cpuMatrix(){
//   int m1[10][10], m2[10][10], sum[10][10];
//   int m = 10;
//   int n = 10;
//
//   int (*a)[m][n];
//   int (*b)[m][n];
//
//   printf("First matrix\n");
//   for (int i = 0; i < m; i++) {
//     for (int j = 0; j < n; j++) {
//       m1[m][n]= (rand() % 9) + 1;
//       printf("%d  ", m1[m][n]);
//
//     }
//     printf("\n");
//   }
//
//   printf("\nSecond matrix\n");
//   for (int i = 0; i < m; i++) {
//     for (int j = 0; j < n; j++) {
//       m2[m][n]= (rand() % 9) + 1;
//       printf("%d  ", m2[m][n]);
//
//     }
//     printf("\n");
//   }
//
//   a = &m1;
//   b = &m2;
//
// }


// void print_matrix(int *m, int n){
// 	for(int i = 0; i < n; i++){
// 		printf("%d\t", m[i]);
// 		if (i % n == n-1)
// 		{
// 			printf("\n");
// 		}
// 	}
//
// 	printf("\n");
// }

void fillMatrixGpu(int *m, int n){
  for (int i = 0; i < n; i++) {
          m[i] = rand()%9+1;
  }
}
int main(){
  int dim = 4;
  int threadsPerBlock = 4;
  int *m1, *m2, *m3;
  int *d_m1, *d_m2, *d_m3;
  int size = dim*dim*sizeof(int);

  // allocate dev
  hipMalloc((void**)&d_m1, size);
  hipMalloc((void**)&d_m2, size);
  hipMalloc((void**)&d_m3, size);

  // allocate host
  m1 = (int*)malloc(size);
  m2 = (int*)malloc(size);
  m3 = (int*)malloc(size);

  // run clock()
  clock_t timeOnGpu = clock();

  //fillMatrix
  fillMatrixGpu(m1, dim*dim);
          fillMatrixGpu(m2, dim*dim);

  for(int i = 0; i < dim; i++){
     for(int j = 0; j < dim; j++){
        printf("%d\t",  m1[i + j * dim]);
          }
        printf("\n");
        }
        printf("\n");
  for(int i = 0; i < dim; i++){
     for(int j = 0; j < dim; j++){
                        printf("%d\t",  m2[i + j* dim]);
          }
        printf("\n");
        }
        printf("\n");
  //host to device
  hipMemcpy(d_m1, m1, size, hipMemcpyHostToDevice);
  hipMemcpy(d_m2, m2, size, hipMemcpyHostToDevice);

  // Kernel function
  gpuMatrix<<<dim*dim/threadsPerBlock, threadsPerBlock>>> (d_m1, d_m2, d_m3, dim);
  //device to host
  hipMemcpy(m3, d_m3, size, hipMemcpyDeviceToHost);

  //print result and time
  for(int i = 0; i < dim; i++){
     for(int j = 0; j < dim; j++){
        printf("%d\t",  m3[i + j * dim]);
          }
        printf("\n");
        }
  printf("time on GPU %f \n", ((double)clock() - timeOnGpu)/CLOCKS_PER_SEC);

  //free memory
  hipFree(d_m1);
  hipFree(d_m2);
  hipFree(d_m3);

  free(m1);
  free(m2);
  free(m3);

  // clock_t timeOnCpu = clock();
  // cpuMatrix();
  // printf("time on CPU %f \n", ((double)clock() - timeOnCpu)/CLOCKS_PER_SEC);

  return 0;
}
